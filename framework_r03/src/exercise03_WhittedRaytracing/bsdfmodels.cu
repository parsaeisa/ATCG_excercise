#include "hip/hip_runtime.h"
#include "bsdfmodels.cuh"

#include "opg/scene/utility/interaction.cuh"

#include <optix.h>

// Schlick's approximation to the fresnel reflectance term
// See https://en.wikipedia.org/wiki/Schlick%27s_approximation
__device__ float fresnel_schlick( const float F0, const float VdotH )
{
    return F0 + ( 1.0f - F0 ) * glm::pow( glm::max(0.0f, 1.0f - VdotH), 5.0f );
}

__device__ glm::vec3 fresnel_schlick( const glm::vec3 F0, const float VdotH )
{
    return F0 + ( glm::vec3(1.0f) - F0 ) * glm::pow( glm::max(0.0f, 1.0f - VdotH), 5.0f );
}


extern "C" __device__ BSDFEvalResult __direct_callable__opaque_evalBSDF(const SurfaceInteraction &si, const glm::vec3 &outgoing_ray_dir, BSDFComponentFlags component_flags)
{
    const OpaqueBSDFData *sbt_data = *reinterpret_cast<const OpaqueBSDFData **>(optixGetSbtDataPointer());

    float NdotV = glm::dot(si.normal, -si.incoming_ray_dir); // incoming_ray_dir points towards surface
    float NdotL = glm::dot(si.normal, outgoing_ray_dir);

    // if (sign(NdotL) == sign(NdotV))
    //    clampedNdotL = abs(NdotL);
    // else
    //    clampedNdotL = 0;
    float clampedNdotL = glm::max(0.0f, NdotL * glm::sign(NdotV));

    glm::vec3 diffuse_bsdf = sbt_data->diffuse_color / M_PIf * clampedNdotL;

    BSDFEvalResult result;
    result.bsdf_value = diffuse_bsdf;
    result.sampling_pdf = 0; // No diffuse BSDF importance sampling
    return result;
}

extern "C" __device__ BSDFSamplingResult __direct_callable__opaque_sampleBSDF(const SurfaceInteraction &si, BSDFComponentFlags component_flags, PCG32 &unused_rng)
{
    const OpaqueBSDFData *sbt_data = *reinterpret_cast<const OpaqueBSDFData **>(optixGetSbtDataPointer());

    BSDFSamplingResult result;
    result.sampling_pdf = 0; // invalid sample

    // Check if there is no specular component present
    if (!has_flag(component_flags, BSDFComponentFlag::IdealReflection))
        return result;
    // Check if the specular component is zero
    if (glm::dot(sbt_data->specular_F0, sbt_data->specular_F0) < 1e-6)
        return result;

    /* Implement:
     * - Specular reflections on opaque materials (BRDF of a specular reflection with given reflectance at normal incidence).
     *   - Compute the outgoing ray direction
     *   - Compute the BSDF for the reflection of the incoming ray direction to the outgoing ray direction.
     *   - Set the sampling pdf to 1 to indicate a valid result (The sampling pdf is used later for stochastic sampling methods)
     */


    return result;
}


extern "C" __device__ BSDFEvalResult __direct_callable__refractive_evalBSDF(const SurfaceInteraction &si, const glm::vec3 &outgoing_ray_dir, BSDFComponentFlags component_flags)
{
    // No direct illumination on refractive materials!
    BSDFEvalResult result;
    result.bsdf_value = glm::vec3(0);
    result.sampling_pdf = 0;
    return result;
}

extern "C" __device__ BSDFSamplingResult __direct_callable__refractive_sampleBSDF(const SurfaceInteraction &si, BSDFComponentFlags component_flags, PCG32 &unused_rng)
{
    const RefractiveBSDFData *sbt_data = *reinterpret_cast<const RefractiveBSDFData **>(optixGetSbtDataPointer());

    BSDFSamplingResult result;
    result.sampling_pdf = 0; // invalid sample

    /* Implement:
     * - Reflections and transmissions on refractive materials.
     *   - Compute the outgoing ray direction.
     *     Hint: Check for `component_flags == +BSDFComponentFlag::IdealReflection` or `component_flags == +BSDFComponentFlag::IdealTransmission`
     *           to determine if a reflection or transmission ray should be generated.
     *           The `+` is neccessary to convert from the `enum` type to `uint32_t`...
     *   - Compute the BSDF for the reflection of the incoming ray direction to the outgoing ray direction.
     *   - Set the sampling pdf to 1 to indicate a valid result (The sampling pdf is used later for stochastic sampling methods).
     *   Hint: The surface normals point outwards.
     *   Hint: You can use Schlick's approximation for the Fresnel term to compute the amount of light reflected or transmitted.
     */


    return result;
}
